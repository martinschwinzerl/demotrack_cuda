#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <chrono>
#include <iostream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "definitions.h"
#include "particle.h"
#include "beam_elements.h"
#include "beamfields.h"
#include "fodo_lattice.h"

__global__ void Track_particles_until_turn(
    demotrack::Particle* particle_set,
    demotrack::int64_type const num_particles,
    double const* __restrict__ lattice_buffer,
    demotrack::uint64_type const max_lattice_buffer_index,
    demotrack::int64_type const until_turn )
{
    namespace dt = demotrack;

    dt::int64_type const STRIDE = blockDim.x * gridDim.x;
    dt::int64_type idx = threadIdx.x + blockIdx.x * blockDim.x;

    for( ; idx < num_particles ; idx += STRIDE )
    {
        dt::Particle* __restrict__ p = &particle_set[ idx ];
        dt::uint64_type const start_at_element = p->at_element;

        while( ( p->state == 1 ) && ( p->at_turn < until_turn ) )
        {
            dt::uint64_type slot_idx = 0;

            while( ( p->state == 1 ) && ( slot_idx < max_lattice_buffer_index ) )
            {
                /* all elements are stored with their type_id as the first
                 * data member -> retrieve this number and dispatch
                 * the track method accordingly */

                dt::beam_element_type const type_id = ( dt::beam_element_type )(
                    int )lattice_buffer[ slot_idx ];

                switch( type_id )
                {
                    case dt::BEAM_ELEMENT_DRIFT: // cf. beam_elements.h
                    {
                        const dt::Drift *const __restrict__ elem =
                            ( dt::Drift const* )&lattice_buffer[ slot_idx ];

                        dt::uint64_type const next_slot_idx =
                            elem->track( *p, slot_idx );

                        dt::Drift::GLOBAL_APERTURE_CHECK( *p );
                        slot_idx = next_slot_idx;
                        break;
                    }

                    case dt::BEAM_ELEMENT_MULTIPOLE: // cf. beam_elements.h
                    {
                        const dt::Multipole *const __restrict__ elem =
                            ( dt::Multipole const* )&lattice_buffer[ slot_idx ];

                        dt::uint64_type const next_slot_idx =
                            elem->track( *p, slot_idx );

                        slot_idx = next_slot_idx;
                        ++p->at_element;
                        break;
                    }

                    case dt::BEAM_ELEMENT_CAVITY: // cf. beam_elements.h
                    {
                        const dt::Cavity *const __restrict__ elem =
                            ( dt::Cavity const* )&lattice_buffer[ slot_idx ];

                        dt::uint64_type const next_slot_idx =
                            elem->track( *p, slot_idx );

                        slot_idx = next_slot_idx;
                        ++p->at_element;
                        break;
                    }

                    #if defined( DEMOTRACK_ENABLE_BEAMFIELDS ) && \
                        DEMOTRACK_ENABLE_BEAMFIELDS == 1

                    case dt::BEAM_ELEMENT_SC_COASTING: // cf. beamfields.h
                    {
                        const dt::SpaceChargeCoasting *const __restrict__ elem =
                            ( dt::SpaceChargeCoasting const* )&lattice_buffer[ slot_idx ];

                        dt::uint64_type const next_slot_idx =
                            elem->track( *p, slot_idx );

                        ++p->at_element;
                        slot_idx = next_slot_idx;
                        break;
                    }

                    #endif /* beamfields enabled */

                    default:
                    {
                        /* unknown beam element -> loose particle and quit */
                        p->state = 0;
                        slot_idx = max_lattice_buffer_index;
                    }
                };

            }

            if( p->state == 1 )
            {
                p->at_element = start_at_element;
                ++p->at_turn;
            }
        }
    }
}

int main( int argc, char* argv[] )
{
    namespace dt = demotrack;

    /* ********************************************************************* */
    /* Prepare particle set to track */

    dt::uint64_type NUM_PARTICLES = 50 * 1024;
    dt::int64_type  TRACK_UNTIL_TURN = 1000;

    if( argc >= 2 )
    {
        NUM_PARTICLES = std::stoi( argv[ 1 ] );

        if( argc >= 3 )
        {
            TRACK_UNTIL_TURN = std::stoi( argv[ 2 ] );
        }
    }
    else
    {
        std::cout << "Usage : " << argv[ 0 ]
                  << " [NUM_PARTICLES] [TRACK_UNTIL_TURN]\r\n"
                  << std::endl;
    }

    double const P0_C    = 470e9;  /* Kinetic energy, [eV]  */
    double const MASS0   = 938.272081e6; /* Proton rest mass, [eV] */
    double const CHARGE0 = 1.0; /* Reference particle charge; here == proton */
    double const DELTA   = 0.0; /* momentum deviation from reference particle */

    std::vector< dt::Particle > particles_host( NUM_PARTICLES );

    dt::uint64_type particle_id = 0u;
    for( auto& p : particles_host )
    {
        p.init( MASS0, CHARGE0, P0_C, DELTA );
        p.id = particle_id++;
    }

    /* ********************************************************************* */
    /* Prepare lattice / machine description: */

    double simple_fodo_lattice[ 64 ];

    /* see fodo_lattice.h for the implementation of create_fodo_lattice */
    dt::uint64_type const LATTICE_SIZE =
        dt::create_fodo_lattice( &simple_fodo_lattice[ 0 ], 64u );

    /* ********************************************************************** */
    /* Allocate buffers on the device */

    dt::Particle* particles_dev = nullptr;
    double* lattice_dev = nullptr;

    auto status = ::hipMalloc( ( void** )&particles_dev,
        sizeof( dt::Particle ) * NUM_PARTICLES );
    assert( status == hipSuccess );

    status = ::hipMalloc( ( void** )&lattice_dev,
        LATTICE_SIZE * sizeof( double ) );
    assert( status == hipSuccess );

    /* Copy particle and lattice data to device */

    status = ::hipMemcpy( lattice_dev, &simple_fodo_lattice[ 0 ],
        LATTICE_SIZE * sizeof( double ), ::hipMemcpyHostToDevice );
    assert( status == hipSuccess );

    status = ::hipMemcpy( particles_dev, particles_host.data(),
        particles_host.size() * sizeof( dt::Particle ),
            ::hipMemcpyHostToDevice );

    assert( status == hipSuccess );

    /* ******************************************************************** */
    /* Estimate block size */

    int BLOCK_SIZE = 0;
    int MIN_GRID_SIZE = 0;

    status = ::hipOccupancyMaxPotentialBlockSize(
        &MIN_GRID_SIZE, /* -> minimum grid size needed for max occupancy */
        &BLOCK_SIZE, /* -> estimated optimal block size */
        Track_particles_until_turn, /* the kernel */
        0u, /* -> dynamic shared memory per block required [bytes] */
        0u /* -> max block size limit for the kernel; 0 == no limit */ );

    assert( status == hipSuccess );

    assert( BLOCK_SIZE > 0 );
    int const GRID_SIZE = ( NUM_PARTICLES + BLOCK_SIZE - 1 ) / BLOCK_SIZE;

    /* ******************************************************************** */
    /* Run kernel: */

    ::hipDeviceProp_t props;
    int device = 0;
    status = ::hipGetDevice( &device );
    assert( status == hipSuccess );

    status = ::hipGetDeviceProperties( &props, device );
    assert( status == hipSuccess );

    char pci_bus_id_str[] =
    {
        '\0', '\0', '\0', '\0', '\0', '\0', '\0', '\0',
        '\0', '\0', '\0', '\0', '\0', '\0', '\0', '\0',
        '\0', '\0', '\0', '\0', '\0', '\0', '\0', '\0',
        '\0', '\0', '\0', '\0', '\0', '\0', '\0', '\0'
    };

    status = ::hipDeviceGetPCIBusId( pci_bus_id_str, 32, device );
    assert( status == hipSuccess );

    std::cout << "number of particles : " << NUM_PARTICLES << "\r\n"
              << "number of turns     : " << TRACK_UNTIL_TURN << "\r\n";

    #if defined( DEMOTRACK_ENABLE_BEAMFIELDS ) && DEMOTRACK_ENABLE_BEAMFIELDS == 1
    std::cout << "space-charge enabled: true\r\n";
    #else
    std::cout << "space-charge enabled: false\r\n";
    #endif /* SC emabled */

    std::cout << "DEVICE              : " << pci_bus_id_str
              << " (" << props.name << " )\r\n"
              << "NUM_OF_BLOCKS       : " << GRID_SIZE << "\r\n"
              << "THREADS_PER_BLOCK   : " << BLOCK_SIZE << "\r\n";

    auto start_time = std::chrono::steady_clock::now();
    Track_particles_until_turn<<< GRID_SIZE, BLOCK_SIZE >>>(
        particles_dev, NUM_PARTICLES, lattice_dev, LATTICE_SIZE,
            TRACK_UNTIL_TURN );
    status = ::hipDeviceSynchronize();

    auto stop_time = std::chrono::steady_clock::now();
    assert( status == hipSuccess );

    std::chrono::duration< double > const wtime = stop_time - start_time;

    std::cout << "-------------------------------------------------------\r\n"
              << "Elapsed time        : " << wtime.count() << " sec total \r\n"
              << "                    : " << wtime.count() / (
                std::max( NUM_PARTICLES * TRACK_UNTIL_TURN,
                          dt::uint64_type{ 1 } ) ) << " sec / particle / turn\r\n"
              << std::endl;

    /* Fetch data */

    status = ::hipMemcpy( particles_host.data(), particles_dev,
                           particles_host.size() * sizeof( dt::Particle ),
                           ::hipMemcpyDeviceToHost );
    assert( status == hipSuccess );

    /* ********************************************************************* */
    /* Verify tracking results */

    dt::uint64_type num_active_particles = 0u;
    dt::uint64_type num_lost_particles = 0u;

    for( auto& p : particles_host )
    {
        if( ( p.state == 1 ) && ( p.at_turn == TRACK_UNTIL_TURN ) )
        {
            ++num_active_particles;
        }
        else if( ( p.state == 0 ) && ( p.at_turn < TRACK_UNTIL_TURN ) )
        {
            ++num_lost_particles;
        }
        else
        {
            std::cerr << "illegal particle id = " << p.id
                      << ", at_turn = " << p.at_turn
                      << ", at_element = " << p.at_element
                      << ", state = " << p.state << std::endl;
        }
    }

    std::cout << "-------------------------------------------------------\r\n"
              << "num lost particles    : " << num_lost_particles << "\r\n"
              << "num active particles  : " << num_active_particles << "\r\n"
              << std::endl;

    /* ********************************************************************* */
    /* Cleaning up, Freeing resources */

    ::hipFree( lattice_dev );
    lattice_dev = nullptr;

    ::hipFree( particles_dev );
    particles_dev = nullptr;

    return 0;
}

